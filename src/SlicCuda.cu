#include "hip/hip_runtime.h"

#include "SlicCudaHost.h"
#include "SlicCudaDevice.h"

// Texture/surface ref can only be declared in one unique file
// This is why we need this files organisation
extern texture<uchar4, hipTextureType2D, hipReadModeElementType> texFrameBGRA;
extern surface<void, cudaSurfaceType2D> surfFrameLab;
extern surface<void, cudaSurfaceType2D> surfLabels;

#include "SlicCudaHost.hcu"
#include "SlicCudaDevice.dcu"


